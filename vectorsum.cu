#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <time.h>
#include <unistd.h>

#define N 1000000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
	int stride = blockDim.x;
	int offset = threadIdx.x * stride;

	for(int i = offset; i < offset+stride; i++){
		out[i] = a[i] + b[i];
	}
}

int main(){
	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	// Allocate host memory
	a = (float*)malloc(sizeof(float) * N);
	b = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);

	// Initialize host arrays
	for(int i = 0; i < N; i++){
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	// Allocate device memory
	hipMalloc((void**)&d_a, sizeof(float) * N);
	hipMalloc((void**)&d_b, sizeof(float) * N);
	hipMalloc((void**)&d_out, sizeof(float) * N);

	// Transfer data from host to device memory
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	// Executing kernel
	clock_t begin = clock();
	vector_add<<<(N+255)/256, 256>>>(d_out, d_a, d_b, N);
	//cudaDeviceSynchronize();
	clock_t time = clock() - begin;
	printf("Time %ld\n", time);


	// Transfer data back to host memory
	hipMemcpy(out, d_out, sizeof(float) * N,
	hipMemcpyDeviceToHost);
	
	// Verification
	for(int i = 0; i < N; i++){
		//printf("%f", out[i]);
		if(fabs(out[i] - a[i] - b[i]) < MAX_ERR){
			printf("%i", i);
			break;
		}
		//assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
	}
	printf("PASSED\n");

	// Deallocate device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);
}
